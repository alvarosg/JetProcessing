/**
 * @file   cudageneralpurpose.cu
 * @author �lvaro S�nchez Gonz�lez <alvarosg@usal.es>
 * @date   Mon Jul  23 2012
 *
 * Copyright (c) 2012 �lvaro S�nchez Gonz�lez
 *
 * @brief  Implementaci�n de la cabecera cudageneralpurpose.h. Este fichero se compila con el compilador de CUDA.
 *
 */
#include "cudageneralpurpose.h"


bool IsCudaAvailable(){
    int number;
    if(CudaSafe(hipGetDeviceCount(&number),"hipGetDeviceCount") == false || number<1){
        return false;
    }
    return true;
}
