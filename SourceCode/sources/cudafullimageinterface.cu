#include "hip/hip_runtime.h"
/**
 * @file   cudafullimageinterface.cu
 * @author �lvaro S�nchez Gonz�lez <alvarosg@usal.es>
 * @date   Mon Jul  23 2012
 *
 * Copyright (c) 2012 �lvaro S�nchez Gonz�lez
 *
 * @brief  Implementaci�n de la cabecera cudafullimageinterface.h. Este fichero se compila con el compilador de CUDA.
 *
 */
#include "cudafullimageinterface.h"


//Kernels

extern "C"
/** Kernel que se ejecuta en la tarjeta gr�fica para obtener el valor absoluto de la imagen compleja.
    @param source Buffer en la memoria de la GPU con la imagen compleja alineada fila tras fila.
    @param result Zona de memoria para el resultado.
    @param sizeX Ancho en p�xeles de la imagen compleja.
    @param sizeY Alto en p�xeles de la imagen compleja.*/
__global__ void AbsoluteValueKernel(complex * source, float * result,int sizeX,int sizeY){
    float real,imag;
    int x = threadIdx.x+blockIdx.x*blockDim.x;
    int y = threadIdx.y+blockIdx.y*blockDim.y;

    if (x<sizeX && y<sizeY){
        real=source[x+sizeX*y].x;
        imag=source[x+sizeX*y].y;
        result[x+sizeX*y] = sqrtf(real*real+imag*imag);
    }
}

/** Kernel que se ejecuta en la tarjeta gr�fica para aplicar una m�scara a la imagen compleja.
    @param source Buffer en la memoria de la GPU con la imagen compleja alineada fila tras fila.
    @param sizeX Ancho en p�xeles de la imagen compleja.
    @param sizeY Alto en p�xeles de la imagen compleja.
    @param x1 Posici�n horizontal de la esquina superior izquierda de la m�scara.
    @param x2 Posici�n horizontal de la esquina inferior derecha de la m�scara.
    @param y1 Posici�n vertical de la esquina superior izquierda de la m�scara.
    @param y2 Posici�n vertical de la esquina inferior derecha de la m�scara.*/
__global__ void ApplyMaskKernel(complex * source,int sizeX,int sizeY,int x1,int y1,int x2, int y2){
    int x = threadIdx.x+blockIdx.x*blockDim.x;
    int y = threadIdx.y+blockIdx.y*blockDim.y;

    if (x<sizeX && y<sizeY && (x<x1 || x>x2 || y<y1 || y>y2)){
        source[x+sizeX*y].x=0;
        source[x+sizeX*y].y=0;
    }
}

/** Kernel que recoloca la imagen compleja tras haber realizado una transformada de fourier.
    @param source Buffer en la memoria de la GPU con la imagen compleja alineada fila tras fila.
    @param result Zona de memoria para el resultado.
    @param sizeX Ancho en p�xeles de la imagen compleja.
    @param sizeY Alto en p�xeles de la imagen compleja.*/
__global__ void ReArrangeFTKernel(complex * source, complex * result,int sizeX,int sizeY){

    int x = threadIdx.x+blockIdx.x*blockDim.x;
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    int xAux,yAux;

    if(x<sizeX/2)
        xAux=x+sizeX/2;
    else
        xAux=x-sizeX/2;
    if(y<sizeY/2)
        yAux=y+sizeY/2;
    else
        yAux=y-sizeY/2;


    if (x<sizeX && y<sizeY && xAux<sizeX && yAux<sizeY ){
        result[xAux+sizeX*yAux]=source[x+sizeX*y];
    }
}

/** Kernel que extrae la fase de una parte de la imagen compleja delimitada por uan m�scara.
    @param source Buffer en la memoria de la GPU con la imagen compleja alineada fila tras fila.
    @param result Zona de memoria para el resultado.
    @param sizeX Ancho en p�xeles de la imagen compleja.
    @param sizeY Alto en p�xeles de la imagen compleja.
    @param x1 Posici�n horizontal de la esquina superior izquierda de la m�scara.
    @param x2 Posici�n horizontal de la esquina inferior derecha de la m�scara.
    @param y1 Posici�n vertical de la esquina superior izquierda de la m�scara.
    @param y2 Posici�n vertical de la esquina inferior derecha de la m�scara.*/
__global__ void CropPhaseKernel(complex * source,float * result,int sizeX,int sizeY,int x1,int y1,int x2, int y2){

    int x = threadIdx.x+blockIdx.x*blockDim.x;
    int y = threadIdx.y+blockIdx.y*blockDim.y;

    int xAux,yAux;
    float real,imag,phase;

    //S�lo calculo y guardo la fase de los puntos dentro de la m�scara

    if (x>=x1 && y>=y1 && x<=x2 && y<=y2 && x<sizeX && y<sizeY){
        xAux=x-x1;
        yAux=y-y1;
        real=source[x+y*sizeX].x;
        imag=source[x+y*sizeX].y;
        //Paso a modulo 2Pi
        phase=atan2f(imag,real);

        //Poner en el intervalo (0,2PI)
        if (phase<0)
            phase+=2*PI_NUM;

        result[xAux+yAux*(x2-x1+1)]=phase;

    }

}


complex * LoadInGPUCUDA(complex * hostPt,int width,int height){

    complex * devicePt;
    CudaSafe( hipMalloc((void **)&devicePt,sizeof(complex)*height*width),"hipMalloc") ;
    CudaSafe( hipMemcpy(devicePt, hostPt, sizeof(complex)*height*width,hipMemcpyHostToDevice),"hipMemcpy");

    return devicePt;
}

complex * MemCpyCUDA(complex * devicePt,int width,int height){
    complex * returnDevicePt;
    CudaSafe( hipMalloc((void **)&returnDevicePt,sizeof(complex)*height*width),"hipMalloc") ;
    CudaSafe( hipMemcpy(returnDevicePt, devicePt, sizeof(complex)*height*width,hipMemcpyDeviceToDevice),"hipMemcpy");
    return returnDevicePt;
}

void FreeCUDA(complex * devicePt){

    CudaSafe( hipFree(devicePt),"hipFree");
}


void GetBufferBackCUDA(complex * hostPt, complex * devicePt, int width,int height){

    CudaSafe( hipMemcpy(hostPt, devicePt, sizeof(complex)*height*width, hipMemcpyDeviceToHost),"hipMemcpy");

}

void ApplyMaskCUDA(complex *devicePt,int width,int height,int x1,int y1,int x2, int y2){
    int blocksX=(15+width)/16;
    int blocksY=(15+height)/16;

    dim3 blocks(blocksX,blocksY);
    dim3 threads(16,16);
    ApplyMaskKernel <<< blocks, threads >>> (devicePt,width,height,x1,y1,x2,y2);

}



void FFTCUDA(complex * devicePt,int width,int height, int mode){
    hipfftHandle plan;

    CudaSafeFFT(hipfftPlan2d(&plan, height, width, HIPFFT_C2C),"hipfftPlan2d");

    if (mode==FORWARD)
        CudaSafeFFT(hipfftExecC2C(plan, (hipfftComplex *)devicePt, (hipfftComplex *)devicePt, HIPFFT_FORWARD),"hipfftExecC2C");
    else if (mode==INVERSE)
        CudaSafeFFT(hipfftExecC2C(plan, (hipfftComplex *)devicePt, (hipfftComplex *)devicePt, HIPFFT_BACKWARD),"hipfftExecC2C");

    CudaSafeFFT(hipfftDestroy(plan),"hipfftDestroy");

}

complex * ReArrangeFTCUDA(complex * devicePt,int width,int height){
    complex * resultDevice;

    CudaSafe( hipMalloc((void **)&resultDevice,sizeof(complex)*height*width),"hipMalloc");
    //Con ese truco consigo un redondeo hacia arriba en la divisi�n
    int blocksX=(15+width)/16;
    int blocksY=(15+height)/16;

    dim3 blocks(blocksX,blocksY);
    dim3 threads(16,16);
    ReArrangeFTKernel <<< blocks, threads >>> (devicePt, resultDevice,width,height);
    CudaSafeKernel("ReArrangeFT");

    return resultDevice;

}

float * GetAbsoluteValueCUDA(complex *devicePt,int width,int height){

    float * resultHost, * resultDevice;
    resultHost=(float *)malloc(sizeof(float)*height*width);
    CudaSafe( hipMalloc((void **)&resultDevice,sizeof(float)*height*width),"hipMalloc");
    //Con ese truco consigo un redondeo hacia arriba en la divisi�n
    int blocksX=(15+width)/16;
    int blocksY=(15+height)/16;

    dim3 blocks(blocksX,blocksY);
    dim3 threads(16,16);
    AbsoluteValueKernel <<< blocks, threads >>> (devicePt, resultDevice,width,height);
    CudaSafeKernel("AbsoluteValue");

    CudaSafe( hipMemcpy(resultHost, resultDevice, sizeof(float)*height*width, hipMemcpyDeviceToHost),"hipMemcpy");
    CudaSafe( hipFree(resultDevice),"hipFree");

    return resultHost;
}


float * CropPhaseCUDA(complex *devicePt,int width,int height,int x1,int y1,int x2, int y2){
    float * resultDevice;

    CudaSafe( hipMalloc((void **)&resultDevice,sizeof(float)*(x2-x1+1)*(y2-y1+1)),"hipMalloc");

    int blocksX=(15+width)/16;
    int blocksY=(15+height)/16;

    dim3 blocks(blocksX,blocksY);
    dim3 threads(16,16);
    CropPhaseKernel <<< blocks, threads >>> (devicePt, resultDevice,width,height,x1,y1,x2,y2);
    CudaSafeKernel("CropPhase");

    return resultDevice;

}
