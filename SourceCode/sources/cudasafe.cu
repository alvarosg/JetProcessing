/**
 * @file   cudasafe.cu
 * @author �lvaro S�nchez Gonz�lez <alvarosg@usal.es>
 * @date   Mon Jul  23 2012
 *
 * Copyright (c) 2012 �lvaro S�nchez Gonz�lez
 *
 * @brief  Implementaci�n de la cabecera cudasafe.h. Este fichero se compila con el compilador de CUDA.
 *
 */
#include "cudasafe.h"


bool CudaSafe(hipError_t error, char* message){

    if(error!=hipSuccess) {
        fprintf(stderr,"ERROR: %s : %s\n",message,hipGetErrorString( error ));
        fflush(stderr);
        return false;
    }
    return true;
}

bool CudaSafeKernel(char* message){
    hipError_t error = hipGetLastError();
    if ( hipSuccess != error ){
        fprintf(stderr, "ERROR: %s : %s\n", message , hipGetErrorString( error ) );
        fflush(stderr);
        return false;
    }
    return true;
}

bool CudaSafeFFT(hipfftResult error, char* message){
    if(error!=HIPFFT_SUCCESS) {
        fprintf(stderr,"ERROR: %s : %s\n",message,cuFFTGetErrorString( error ));
        fflush(stderr);
        return false;
    }
    return true;

}

char * cuFFTGetErrorString(hipfftResult error){
    char * errorS;
    errorS=(char *)malloc(30 *sizeof(char));
    switch(error){
        case HIPFFT_SUCCESS: strcpy(errorS,"HIPFFT_SUCCESS"); break;
        case HIPFFT_INVALID_PLAN: strcpy(errorS,"HIPFFT_INVALID_PLAN"); break;
        case HIPFFT_ALLOC_FAILED: strcpy(errorS,"HIPFFT_ALLOC_FAILED"); break;
        case HIPFFT_INVALID_TYPE: strcpy(errorS,"HIPFFT_INVALID_TYPE"); break;
        case HIPFFT_INVALID_VALUE: strcpy(errorS,"HIPFFT_INVALID_VALUE"); break;
        case HIPFFT_INTERNAL_ERROR: strcpy(errorS,"HIPFFT_INTERNAL_ERROR"); break;
        case HIPFFT_EXEC_FAILED: strcpy(errorS,"HIPFFT_EXEC_FAILED"); break;
        case HIPFFT_SETUP_FAILED: strcpy(errorS,"HIPFFT_SETUP_FAILED"); break;
        case HIPFFT_INVALID_SIZE: strcpy(errorS,"HIPFFT_INVALID_SIZE"); break;
        default: strcpy(errorS,"UNKNOWN");
    }

    return errorS;

}
