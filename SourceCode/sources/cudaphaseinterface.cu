#include "hip/hip_runtime.h"
/**
 * @file   cudaphaseinterface.cu
 * @author �lvaro S�nchez Gonz�lez <alvarosg@usal.es>
 * @date   Mon Jul  23 2012
 *
 * Copyright (c) 2012 �lvaro S�nchez Gonz�lez
 *
 * @brief  Implementaci�n de la cabecera cudaphaseinterface.h. Este fichero se compila con el compilador de CUDA.
 *
 */
#include "cudaphaseinterface.h"


//Kernels

extern "C"

/** Kernel que se ejecuta en la tarjeta gr�fica como primer paso para escanear la fase.
    @param source Buffer en la memoria de la GPU con la imagen de la fase alineada fila tras fila.
    @param accumulator Zona de memoria que contiene el n�mero de saltos de fase encontrados hasta llegar a cada punto.
    @param sizeX Ancho en p�xeles de la imagen.
    @param sizeY Alto en p�xeles de la imagen.*/
__global__ void ScanPhaseStep1Kernel(float * source,int * accumulator, int sizeX,int sizeY){
    int i = threadIdx.x;
    int x;
    int y=(sizeY/2);
    //El hilo n�mero 0 asigna al punto central un offset de 0
    if(i==0)
        accumulator[sizeX/2+sizeX*y]=0;

    //Sincronizo los hilos
    __syncthreads();

    //Voy calculando secuencialmente por fila intermedia si hay grandes saltos en la fase
    //y en base a ello aumento o decremento el acumulador para cada punto
    //El hilo 0 va hacia la izquierda y el hilo 1 hacia la derecha
    if(i==0){
        for(x=sizeX/2-1;x>=0;x--){
            accumulator[x+sizeX*y]=accumulator[x+1+sizeX*y];

            if      ((source[x+sizeX*y]-source[x+1+sizeX*y])>PI_NUM)
                accumulator[x+sizeX*y]--;

            else if ((source[x+1+sizeX*y]-source[x+sizeX*y])>PI_NUM)
                accumulator[x+sizeX*y]++;
        }
    }
    else if(i==1){
        for(x=sizeX/2+1;x<sizeX;x++){
            accumulator[x+sizeX*y]=accumulator[x-1+sizeX*y];

            if      ((source[x+sizeX*y]-source[x-1+sizeX*y])>PI_NUM)
                accumulator[x+sizeX*y]--;

            else if ((source[x-1+sizeX*y]-source[x+sizeX*y])>PI_NUM)
                accumulator[x+sizeX*y]++;
        }
    }
}

/** Kernel que se ejecuta en la tarjeta gr�fica como segundo paso para escanear la fase.
    @param source Buffer en la memoria de la GPU con la imagen de la fase alineada fila tras fila.
    @param accumulator Zona de memoria que contiene el n�mero de saltos de fase encontrados hasta llegar a cada punto.
    @param sizeX Ancho en p�xeles de la imagen.
    @param sizeY Alto en p�xeles de la imagen.*/
__global__ void ScanPhaseStep2Kernel(float * source,int * accumulator, int sizeX,int sizeY){

    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y;

    //A partir de los datos de la l�nea central, voy calculando hacia arriba(bloques de �ndice menos que el ancho)
    //y hacia abajo (bloques de �ndice mayor que el ancho), en paralelo para
    //cada linea vertical y direcci�n, los saltos de fase

    //Detecto la direcci�n y asigno la columna para los bloques cuyo �ndice est� en la segunda mitad
    //Hacia arriba
    if(x < (2*sizeX)){
        int dir=x/sizeX;
        x=x%sizeX;
        if(dir==0){
            for(y=sizeY/2-1;y>=0;y--){
                accumulator[x+y*sizeX]=accumulator[x+(y+1)*sizeX];

                if      ((source[x+y*sizeX]-source[x+(y+1)*sizeX])>PI_NUM)
                    accumulator[x+y*sizeX]--;

                else if ((source[x+(y+1)*sizeX]-source[x+y*sizeX])>PI_NUM)
                    accumulator[x+y*sizeX]++;
            }
        }
        //Hacia abajo
        else if(dir==1){
            for(y=sizeY/2+1;y<sizeY;y++){
                accumulator[x+y*sizeX]=accumulator[x+(y-1)*sizeX];

                if      ((source[x+y*sizeX]-source[x+(y-1)*sizeX])>PI_NUM)
                    accumulator[x+y*sizeX]--;

                else if ((source[x+(y-1)*sizeX]-source[x+y*sizeX])>PI_NUM)
                    accumulator[x+y*sizeX]++;
            }
        }
    }
}

/** Kernel que se ejecuta en la tarjeta gr�fica como tercer paso para escanear la fase.
    @param source Buffer en la memoria de la GPU con la imagen de la fase alineada fila tras fila.
    @param accumulator Zona de memoria que contiene el n�mero de saltos de fase encontrados hasta llegar a cada punto.
    @param sizeX Ancho en p�xeles de la imagen.
    @param sizeY Alto en p�xeles de la imagen.*/
__global__ void ScanPhaseStep3Kernel(float * source,int * accumulator, int sizeX,int sizeY){
    int x = threadIdx.x+blockIdx.x*blockDim.x;
    int y = threadIdx.y+blockIdx.y*blockDim.y;

    //Sumo m�ltiplos de 2Pi en funci�n de la fase acumulada

    if(x<sizeX && y<sizeY){

        source[x+y*sizeX]=source[x+y*sizeX]+((float)(accumulator[x+y*sizeX]))*2*PI_NUM;
    }
}

/** Kernel que se ejecuta en la tarjeta gr�fica para restar la fase de la referencia.
    @param source Buffer en la memoria de la GPU con la imagen de la fase alineada fila tras fila.
    @param offset Buffer con la imagen de la fase de la referencia alineada fila tras fila.
    @param sizeX Ancho en p�xeles de la imagen.
    @param sizeY Alto en p�xeles de la imagen.*/
__global__ void SubtractKernel(float * source,float * offset, int sizeX,int sizeY){
    int x = threadIdx.x+blockIdx.x*blockDim.x;
    int y = threadIdx.y+blockIdx.y*blockDim.y;

    //Sumo m�ltiplos de 2Pi en funci�n de la fase acumulada

    if(x<sizeX && y<sizeY){

        source[x+y*sizeX]=source[x+y*sizeX]-offset[x+y*sizeX];
    }
}
/** Kernel que se ejecuta en la tarjeta gr�fica para calcular el opuesto de la fase.
    @param source Buffer en la memoria de la GPU con la imagen de la fase alineada fila tras fila.
    @param sizeX Ancho en p�xeles de la imagen.
    @param sizeY Alto en p�xeles de la imagen.*/
__global__ void AdditiveInverseKernel(float * source,int sizeX,int sizeY){
    int x = threadIdx.x+blockIdx.x*blockDim.x;
    int y = threadIdx.y+blockIdx.y*blockDim.y;

    //Sumo m�ltiplos de 2Pi en funci�n de la fase acumulada

    if(x<sizeX && y<sizeY){

        source[x+y*sizeX]=-source[x+y*sizeX];
    }
}

/** Kernel que se ejecuta en la tarjeta gr�fica como primer paso para eliminar la inclinaci�n lineal horizontal usando los bordes.
    @param source Buffer en la memoria de la GPU con la imagen de la fase alineada fila tras fila.
    @param sizeX Ancho en p�xeles de la imagen.
    @param sizeY Alto en p�xeles de la imagen.
    @param avgSlope Puntero a la zona de memoria donde se escribir� la inclinaci�n media.*/
__global__ void RemoveHorizontalLinearTiltBordersStep1Kernel(float * source,int sizeX,int sizeY,float * avgSlope){
    int y;

    //Recorro cada l�nea horizontal calculando la pendiente entre el primer valor y el �ltimo
    (*avgSlope)=0;
    for (y=0;y<sizeY;y++){
        (*avgSlope)+=(source[(sizeX-1)+y*sizeX]-source[y*sizeX])/(sizeX-1);
    }
    (*avgSlope)=(*avgSlope)/sizeY;
}
/** Kernel que se ejecuta en la tarjeta gr�fica como segundo paso para eliminar la inclinaci�n lineal horizontal usando los bordes.
    @param source Buffer en la memoria de la GPU con la imagen de la fase alineada fila tras fila.
    @param sizeX Ancho en p�xeles de la imagen.
    @param sizeY Alto en p�xeles de la imagen.
    @param avgSlope Puntero a la zona de memoria con la inclinaci�n media.*/
__global__ void RemoveHorizontalLinearTiltBordersStep2Kernel(float * source,int sizeX,int sizeY,float * avgSlope){
    int x = threadIdx.x+blockIdx.x*blockDim.x;
    int y = threadIdx.y+blockIdx.y*blockDim.y;

    //Aplico a cada punto la correcci�n en funci�n de la inclinaci�n
    if(x<sizeX && y<sizeY){
        source[x+y*sizeX]=source[x+y*sizeX]-(*avgSlope)*(x-sizeX/2);
    }
}

/** Kernel que se ejecuta en la tarjeta gr�fica como primer paso para eliminar la inclinaci�n lineal horizontal.
    @param source Buffer en la memoria de la GPU con la imagen de la fase alineada fila tras fila.
    @param sizeX Ancho en p�xeles de la imagen.
    @param sizeY Alto en p�xeles de la imagen.
    @param slopesArray Puntero a la zona de memoria donde se escribir� la inclinaci�n de cada una de las rectas.*/
__global__ void RemoveHorizontalLinearTiltStep1Kernel(float * source,int sizeX,int sizeY, float * slopesArray){
    int x;
    int y = threadIdx.x+blockIdx.x*blockDim.x;
    //Calculo la pendiente de cada recta horizontal con una regresi�n
    float sumX=0;
    float sumX2=0;
    float sumZ=0;
    float sumXZ=0;
    if(y<sizeY){
        for (x=0;x<sizeX;x++){
            sumX+=x;
            sumX2+=x*x;
            sumZ+=source[x+sizeX*y];
            sumXZ+=source[x+sizeX*y]*x;
        }
        slopesArray[y]=(sumX*sumZ-sizeX*sumXZ)/(sumX*sumX-sizeX*sumX2);
    }
}

/** Kernel que se ejecuta en la tarjeta gr�fica como segundo paso para eliminar la inclinaci�n lineal horizontal.
    @param source Buffer en la memoria de la GPU con la imagen de la fase alineada fila tras fila.
    @param sizeX Ancho en p�xeles de la imagen.
    @param sizeY Alto en p�xeles de la imagen.
    @param avgSlope Puntero a la zona de memoria donde se escribir� la inclinaci�n media.
    @param slopesArray Puntero a la zona de memoria con la inclinaci�n de cada una de las rectas.*/
__global__ void RemoveHorizontalLinearTiltStep2Kernel(float * source,int sizeX,int sizeY,float * avgSlope, float * slopesArray){
    int y;

    //Hago una media de las pendientes
    (*avgSlope)=0;
    for (y=0;y<sizeY;y++){
        (*avgSlope)+=slopesArray[y];
    }
    (*avgSlope)=(*avgSlope)/sizeY;
}

/** Kernel que se ejecuta en la tarjeta gr�fica como tercer paso para eliminar la inclinaci�n lineal horizontal.
    @param source Buffer en la memoria de la GPU con la imagen de la fase alineada fila tras fila.
    @param sizeX Ancho en p�xeles de la imagen.
    @param sizeY Alto en p�xeles de la imagen.
    @param avgSlope Puntero a la zona de memoria con la inclinaci�n media.*/
__global__ void RemoveHorizontalLinearTiltStep3Kernel(float * source,int sizeX,int sizeY,float * avgSlope){
    int x = threadIdx.x+blockIdx.x*blockDim.x;
    int y = threadIdx.y+blockIdx.y*blockDim.y;

    //Aplico a cada punto la correcci�n en funci�n de la inclinaci�n
    if(x<sizeX && y<sizeY){
        source[x+y*sizeX]=source[x+y*sizeX]-(*avgSlope)*(x-sizeX/2);
    }
}

/** Kernel que se ejecuta en la tarjeta gr�fica como primer paso para eliminar la inclinaci�n lineal vertical.
    @param source Buffer en la memoria de la GPU con la imagen de la fase alineada fila tras fila.
    @param sizeX Ancho en p�xeles de la imagen.
    @param sizeY Alto en p�xeles de la imagen.
    @param slopesArray Puntero a la zona de memoria donde se escribir� la inclinaci�n de cada una de las rectas.*/
__global__ void RemoveVerticalLinearTiltStep1Kernel(float * source,int sizeX,int sizeY, float * slopesArray){
    int y;
    int x = threadIdx.x+blockIdx.x*blockDim.x;
    //Calculo la pendiente de cada recta vertical con una regresi�n
    float sumY=0;
    float sumY2=0;
    float sumZ=0;
    float sumYZ=0;
    if(x<sizeX){
        for (y=0;y<sizeY;y++){
            sumY+=y;
            sumY2+=y*y;
            sumZ+=source[x+sizeX*y];
            sumYZ+=source[x+sizeX*y]*y;
        }
        slopesArray[x]=(sumY*sumZ-sizeY*sumYZ)/(sumY*sumY-sizeY*sumY2);
    }
}

/** Kernel que se ejecuta en la tarjeta gr�fica como segundo paso para eliminar la inclinaci�n lineal vertical.
    @param source Buffer en la memoria de la GPU con la imagen de la fase alineada fila tras fila.
    @param sizeX Ancho en p�xeles de la imagen.
    @param sizeY Alto en p�xeles de la imagen.
    @param avgSlope Puntero a la zona de memoria donde se escribir� la inclinaci�n media.
    @param slopesArray Puntero a la zona de memoria con la inclinaci�n de cada una de las rectas.*/
__global__ void RemoveVerticalLinearTiltStep2Kernel(float * source,int sizeX,int sizeY,float * avgSlope, float * slopesArray){
    int x;

    //Hago una media de las pendientes
    (*avgSlope)=0;
    for (x=0;x<sizeX;x++){
        (*avgSlope)+=slopesArray[x];
    }
    (*avgSlope)=(*avgSlope)/sizeX;
}

/** Kernel que se ejecuta en la tarjeta gr�fica como tercer paso para eliminar la inclinaci�n lineal vertical.
    @param source Buffer en la memoria de la GPU con la imagen de la fase alineada fila tras fila.
    @param sizeX Ancho en p�xeles de la imagen.
    @param sizeY Alto en p�xeles de la imagen.
    @param avgSlope Puntero a la zona de memoria con la inclinaci�n media.*/
__global__ void RemoveVerticalLinearTiltStep3Kernel(float * source,int sizeX,int sizeY,float * avgSlope){
    int x = threadIdx.x+blockIdx.x*blockDim.x;
    int y = threadIdx.y+blockIdx.y*blockDim.y;

    //Aplico a cada punto la correcci�n en funci�n de la inclinaci�n
    if(x<sizeX && y<sizeY){
        source[x+y*sizeX]=source[x+y*sizeX]-(*avgSlope)*(y-sizeY/2);
    }
}
/** Kernel que se ejecuta en la tarjeta gr�fica como primer paso para fijar el suelo de la imagen y quitar ruido.
    @param source Buffer en la memoria de la GPU con la imagen de la fase alineada fila tras fila.
    @param sizeX Ancho en p�xeles de la imagen.
    @param sizeY Alto en p�xeles de la imagen.
    @param avgValue Puntero a la zona de memoria conde se escribir� el valor del suelo calculado.*/
__global__ void SetGroundStep1Kernel(float * source,int sizeX,int sizeY,float * avgValue){
    int y;
    //Calculo la media de todos los valores en las rectas verticales laterales de los extremos
    (*avgValue)=0;
    for (y=0;y<sizeY;y++){
        (*avgValue)+=source[y*sizeX]+source[(sizeX-1)+y*sizeX];
    }
    (*avgValue)=(*avgValue)/(2*sizeY);
}

/** Kernel que se ejecuta en la tarjeta gr�fica como segundo paso para fijar el suelo de la imagen y quitar ruido.
    @param source Buffer en la memoria de la GPU con la imagen de la fase alineada fila tras fila.
    @param sizeX Ancho en p�xeles de la imagen.
    @param sizeY Alto en p�xeles de la imagen.
    @param avgValue Puntero a la zona de memoria con el valor del suelo.*/
__global__ void SetGroundStep2Kernel(float * source,int sizeX,int sizeY,float * avgValue){
    int x = threadIdx.x+blockIdx.x*blockDim.x;
    int y = threadIdx.y+blockIdx.y*blockDim.y;

    //Si el valor en el punto es menor que el calculado, asigno el calculado

    if(x<sizeX && y<sizeY){
        if(source[x+y*sizeX] < (*avgValue)){
            source[x+y*sizeX]=(*avgValue);
        }
        source[x+y*sizeX]=source[x+y*sizeX]-(*avgValue);
    }
}

/** Kernel que se ejecuta en la tarjeta gr�fica para a�adir un offset y aplicar una ganancia a la imagen.
    @param source Buffer en la memoria de la GPU con la imagen de la fase alineada fila tras fila.
    @param sizeX Ancho en p�xeles de la imagen.
    @param sizeY Alto en p�xeles de la imagen.
    @param offset Valor del offset a aplicar.
    @param factor Valor de la ganancia a aplicar.*/
__global__ void OffsetAndMultiplyKernel(float * source,int sizeX,int sizeY,float offset,float factor){
    int x = threadIdx.x+blockIdx.x*blockDim.x;
    int y = threadIdx.y+blockIdx.y*blockDim.y;

    //Recalculo el valor en funci�n del offset y el factor de multiplicaci�n

    if(x<sizeX && y<sizeY){
        source[x+y*sizeX]=source[x+y*sizeX]*factor+offset;
    }
}

/** Kernel que se ejecuta en la tarjeta gr�fica como primer paso (Buscar m�ximos) para simetrizar la imagen.
    @param source Buffer en la memoria de la GPU con la imagen de la fase alineada fila tras fila.
    @param sizeX Ancho en p�xeles de la imagen.
    @param sizeY Alto en p�xeles de la imagen.
    @param maxPos Puntero a la zona de memoria donde se excribir�n las posiciones de los m�ximos.*/
__global__ void MakeSymmetricStep1Kernel(float * source,int sizeX,int sizeY,int * maxPos){
    int x;
    int y=threadIdx.x+blockIdx.x*blockDim.x;
    float maxValue;
    //Busco la posici�n del m�ximo en cada recta horizontal
    if(y<sizeY){
        maxPos[y]=0;
        maxValue=source[y*sizeX];

        for (x=1;x<sizeX;x++){
            if (source[x+y*sizeX]> maxValue){
                maxValue=source[x+y*sizeX];
                maxPos[y]=x;
            }
        }
    }
}

/** Kernel que se ejecuta en la tarjeta gr�fica como segundo paso para simetrizar la imagen.
    @param source Buffer en la memoria de la GPU con la imagen de la fase alineada fila tras fila.
    @param dest Puntero a la zona donde se debe escribir la salida.
    @param sizeX Ancho en p�xeles de la imagen.
    @param sizeY Alto en p�xeles de la imagen.
    @param maxPos Puntero a la zona de memoria donde se encuentran las posiciones de los m�ximos.*/
__global__ void MakeSymmetricStep2Kernel(float * source,float * dest,int sizeX,int sizeY,int * maxPos){
    int x = threadIdx.x+blockIdx.x*blockDim.x;
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    float meanValue;
    int dist;

    //Centro el m�ximo y promedio lateralmente respecto del m�ximo
    if(x<sizeX && y<sizeY){
        dist=abs(x-maxPos[y]);
        if((maxPos[y]+dist) < sizeX && (maxPos[y]-dist) >= 0)
            meanValue=(source[(maxPos[y]+dist)+y*sizeX]+source[(maxPos[y]-dist)+y*sizeX])/2;
        else if ((maxPos[y]+dist) < sizeX)
            meanValue=(source[(maxPos[y]+dist)+y*sizeX]+source[0+y*sizeX])/2;
        else if ((maxPos[y]-dist) >=0)
            meanValue=(source[sizeX-1+y*sizeX]+source[(maxPos[y]-dist)+y*sizeX])/2;
        else
            meanValue=0;

        if(sizeX/2+dist < sizeX)
            dest[(sizeX/2+dist)+y*sizeX]=meanValue;
        if(sizeX/2-dist >= 0)
            dest[(sizeX/2-dist)+y*sizeX]=meanValue;



    }
}

/** Kernel que se ejecuta en la tarjeta gr�fica para el c�lculo de la derivada de la imagen.
    @param source Buffer en la memoria de la GPU con la imagen de la fase alineada fila tras fila.
    @param dest Puntero a la zona donde se debe escribir la derivada.
    @param sizeX Ancho en p�xeles de la imagen.
    @param sizeY Alto en p�xeles de la imagen.
    @param dx Distancia en entre dos puntos.
    @param points N�mero de vecinos utilizados para el c�lculo de la derivada.*/
__global__ void DerivateKernel(float * source,float * dest,int sizeX,int sizeY,float dx,int points){
    int x = threadIdx.x+blockIdx.x*blockDim.x;
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    float value=0;
    int count=0;
    int i;

    //Calculo la derivada en cada punto, promediando la pendiente con 2 puntos anteriores y posteriores

    if(x<sizeX && y<sizeY){
        for(i=1;i<=points;i++){
            if(x-i>=0){
                value+=(source[x+y*sizeX]-source[(x-i)+y*sizeX])/i;
                count++;
            }
            if(x+i<sizeX){
                value+=(source[x+i+y*sizeX]-source[x+y*sizeX])/i;
                count++;
            }
        }
        dest[x+y*sizeX]=value/(dx*count);
    }
}
/** Kernel que se ejecuta en la tarjeta gr�fica para el c�lculo de la inversi�n de Abel.
    @param derivative Puntero a la matriz con la derivada en cada punto.
    @param dest Puntero a la zona donde se debe escribir la salida.
    @param sizeX Ancho en p�xeles de la imagen.
    @param sizeY Alto en p�xeles de la imagen.*/
__global__ void AbelInversionKernel(float * derivative,float * dest,int sizeX,int sizeY){

    int x = threadIdx.x+blockIdx.x*blockDim.x;
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    int i;
    float iFloat;
    float xFloat=x-sizeX/2;
    float value=0;

    //Cada hilo calcula la integral en un punto a una distancia determinada

    if(x<sizeX && y<sizeY){
        xFloat=(x-sizeX/2);
        if(xFloat>=0){
            //Calculo el integrando para cada posici�n correspondiente y lo voy sumando al acumulador
            //haciendo aproximaci�n trapezoidal (Sumo la mitad del primero y del �ltimo)
            for(i=x;i<sizeX;i++){
                //Evito la singularidad en i=x, asignandole el siguiente valor
                if(i==x){
                    if(i+1<sizeX){
                        iFloat=(i+1-(int)sizeX/2);
                        value+=-derivative[i+1+y*sizeX]/(PI_NUM*sqrt(iFloat*iFloat-xFloat*xFloat))/2;
                    }
                }
                else if(i==sizeX-1){
                    iFloat=(i-(int)sizeX/2);
                    value+=-derivative[i+y*sizeX]/(PI_NUM*sqrt(iFloat*iFloat-xFloat*xFloat))/2;
                }
                else{
                    iFloat=(i-(int)sizeX/2);
                    value+=-derivative[i+y*sizeX]/(PI_NUM*sqrt(iFloat*iFloat-xFloat*xFloat));
                }
            }
        }
        else{
            //Calculo el integrando para cada posici�n correspondiente y lo voy sumando al acumulador
            //haciendo aproximaci�n trapezoidal (Sumo la mitad del primero y del �ltimo)
            for(i=0;i<=x;i++){
                //Evito la singularidad en i=x, asignandole el siguiente valor
                if(i==x){
                    if(i-1>=0){
                        iFloat=(i-1-(int)sizeX/2);
                        value+=derivative[i-1+y*sizeX]/(PI_NUM*sqrt(iFloat*iFloat-xFloat*xFloat))/2;
                    }
                }
                else if(i==0){
                    iFloat=(i-(int)sizeX/2);
                    value+=derivative[i+y*sizeX]/(PI_NUM*sqrt(iFloat*iFloat-xFloat*xFloat))/2;
                }
                else{
                    iFloat=(i-(int)sizeX/2);
                    value+=derivative[i+y*sizeX]/(PI_NUM*sqrt(iFloat*iFloat-xFloat*xFloat));

                }
            }
        }
        dest[x+y*sizeX]=value;
    }
}

/** Kernel que se ejecuta en la tarjeta gr�fica para la obtenci�n de un rect�ngulo dentro de la imagen de la fase.
    @param source Buffer en la memoria de la GPU con la imagen de la fase alineada fila tras fila.
    @param result Puntero a la zona donde se debe escribir la salida.
    @param sizeX Ancho en p�xeles de la imagen.
    @param sizeY Alto en p�xeles de la imagen.
    @param x1 Posici�n horizontal de la esquina superior izquierda de la m�scara.
    @param x2 Posici�n horizontal de la esquina inferior derecha de la m�scara.
    @param y1 Posici�n vertical de la esquina superior izquierda de la m�scara.
    @param y2 Posici�n vertical de la esquina inferior derecha de la m�scara.*/
__global__ void CropRectangleKernel(float * source,float * result,int sizeX,int sizeY,int x1,int y1,int x2, int y2){

    int x = threadIdx.x+blockIdx.x*blockDim.x;
    int y = threadIdx.y+blockIdx.y*blockDim.y;

    int xAux,yAux;

    //Asigno los puntos adecuadamente

    if (x>=x1 && y>=y1 && x<=x2 && y<=y2 && x<sizeX && y<sizeY){
        xAux=x-x1;
        yAux=y-y1;

        result[xAux+yAux*(x2-x1+1)]=source[x+y*sizeX];
    }
}


/** Kernel que se ejecuta en la tarjeta gr�fica para la obtenci�n de una l�nea de la fase.
    @param source Buffer en la memoria de la GPU con la imagen de la fase alineada fila tras fila.
    @param sizeX Ancho en p�xeles de la imagen.
    @param sizeY Alto en p�xeles de la imagen.
    @param x1 Posici�n horizontal de la esquina superior izquierda de la m�scara.
    @param x2 Posici�n horizontal de la esquina inferior derecha de la m�scara.
    @param y1 Posici�n vertical de la esquina superior izquierda de la m�scara.
    @param y2 Posici�n vertical de la esquina inferior derecha de la m�scara.
    @param output Puntero a la zona donde se debe escribir la salida.
    @param outputSize N�mero de muestras para la recta.*/
__global__ void GetPhaseLineKernel(float * source,int sizeX,int sizeY,int x1,int y1,int x2, int y2,float * output,int outputSize){
    int j=threadIdx.x+blockIdx.x*blockDim.x;

    //Calculo las coordenadas del pixel en la imagen original que tengo que colocar en la posici�n j d ela l�nea
    int x=x1+((x2-x1)*j)/(outputSize-1);
    int y=y1+((y2-y1)*j)/(outputSize-1);

    if(j<outputSize){
        if(x>=0 && y>=0 && x<sizeX &&y < sizeY)
            output[j]=source[x+y*sizeX];
        else
            output[j]=0;
    }

}


float * LoadInGPUCUDA(float * hostPt,int width,int height){

    float * devicePt;
    CudaSafe( hipMalloc((void **)&devicePt,sizeof(float)*height*width),"hipMalloc");
    CudaSafe( hipMemcpy(devicePt, hostPt, sizeof(float)*height*width,hipMemcpyHostToDevice),"hipMemcpy");

    return devicePt;
}

float * MemCpyCUDA(float * devicePt,int width,int height){
    float * returnDevicePt;
    CudaSafe( hipMalloc((void **)&returnDevicePt,sizeof(float)*height*width),"hipMalloc") ;
    CudaSafe( hipMemcpy(returnDevicePt, devicePt, sizeof(float)*height*width,hipMemcpyDeviceToDevice),"hipMemcpy");
    return returnDevicePt;
}



void FreeCUDA(float * devicePt){
    CudaSafe( hipFree(devicePt),"hipFree");
}


void GetBufferBackCUDA(float * hostPt, float * devicePt, int width,int height){

    CudaSafe( hipMemcpy(hostPt, devicePt, sizeof(float)*height*width, hipMemcpyDeviceToHost),"hipMemcpy");
}


void ScanPhaseCUDA(float * devicePt, int width,int height){


    int * phaseAccumulator;

    CudaSafe( hipMalloc((void **)&phaseAccumulator,sizeof(int)*height*width),"hipMalloc");

    int blocksX=(15+width)/16;
    int blocksY=(15+height)/16;
    dim3 blocks(blocksX,blocksY);
    dim3 threads(16,16);

    //Escaneo la l�nea horizontal central, hacia derecha e izquierda (con dos hilos), acumulando saltos de fase
    ScanPhaseStep1Kernel <<< 1, 2>>> (devicePt,phaseAccumulator,width,height);
    CudaSafeKernel("ScanPhaseStep1");
    //A partir de la l�nea central escaneo en cascada cada l�nea vertical, hacia arriba y hacia abajo, acumulando saltos de fase
    ScanPhaseStep2Kernel <<< (int)(2*width+31)/32, 32 >>> (devicePt,phaseAccumulator,width,height);
    CudaSafeKernel("ScanPhaseStep2");
    //Sumo m�ltiplos de 2Pi por cada salto de fase
    ScanPhaseStep3Kernel <<< blocks, threads >>> (devicePt,phaseAccumulator,width,height);
    CudaSafeKernel("ScanPhaseStep3");

    CudaSafe( hipFree(phaseAccumulator),"hipFree");


}


void SubtractCUDA(float * devicePt,float * offsetDevicePt, int width,int height){
    int blocksX=(15+width)/16;
    int blocksY=(15+height)/16;
    dim3 blocks(blocksX,blocksY);
    dim3 threads(16,16);

    SubtractKernel <<< blocks, threads >>> (devicePt,offsetDevicePt,width,height);
    CudaSafeKernel("Subtract");

}

void AdditiveInverseCUDA(float * devicePt, int width,int height){

    int blocksX=(15+width)/16;
    int blocksY=(15+height)/16;
    dim3 blocks(blocksX,blocksY);
    dim3 threads(16,16);

    AdditiveInverseKernel <<< blocks, threads >>> (devicePt,width,height);
    CudaSafeKernel("AdditiveInverse");

}

void RemoveHorizontalLinearTiltBordersCUDA(float * devicePt, int width,int height){

    float * avgSlope;
    CudaSafe( hipMalloc((void **)&avgSlope,sizeof(float)),"hipMalloc");

    int blocksX=(15+width)/16;
    int blocksY=(15+height)/16;
    dim3 blocks(blocksX,blocksY);
    dim3 threads(16,16);

    //Calculo la media de las pendientes entre el primer y el �ltimo punto de cada l�nea vertical
    RemoveHorizontalLinearTiltBordersStep1Kernel <<< 1, 1 >>> (devicePt,width,height,avgSlope);
    CudaSafeKernel("RemoveHorizontalLinearTiltBordersStep1");

    //Aplico la correci�n a cada punto en funci�n de esa pendiente media
    RemoveHorizontalLinearTiltBordersStep2Kernel <<< blocks, threads >>> (devicePt,width,height,avgSlope);
    CudaSafeKernel("RemoveHorizontalLinearTiltBordersStep2");

    CudaSafe( hipFree(avgSlope),"hipFree");

}


void RemoveHorizontalLinearTiltCUDA(float * devicePt, int width,int height){
    float * avgSlope;
    float * slopesArray;
    CudaSafe( hipMalloc((void **)&avgSlope,sizeof(float)),"hipMalloc");
    CudaSafe( hipMalloc((void **)&slopesArray,height*sizeof(float)),"hipMalloc");

    int blocksX=(15+width)/16;
    int blocksY=(15+height)/16;
    dim3 blocks(blocksX,blocksY);
    dim3 threads(16,16);

    //Calculo la pendiente para cada una de las rectas verticales, por un ajuste de m�nimos cuadrados
    RemoveHorizontalLinearTiltStep1Kernel <<< (int)(height+31)/32, 32 >>> (devicePt,width,height,slopesArray);
    CudaSafeKernel("RemoveHorizontalLinearTiltStep1");

    //Calculo la media de todas las pendientes
    RemoveHorizontalLinearTiltStep2Kernel <<< 1, 1 >>> (devicePt,width,height,avgSlope,slopesArray);
    CudaSafeKernel("RemoveHorizontalLinearTiltStep2");

    //Aplico la correci�n a cada punto en funci�n de esa pendiente media
    RemoveHorizontalLinearTiltStep3Kernel <<< blocks, threads >>> (devicePt,width,height,avgSlope);
    CudaSafeKernel("RemoveHorizontalLinearTiltStep3");

    CudaSafe( hipFree(avgSlope),"hipFree");
    CudaSafe( hipFree(slopesArray),"hipFree");

}


void RemoveVerticalLinearTiltCUDA(float * devicePt, int width,int height){
    float * avgSlope;
    float * slopesArray;
    CudaSafe( hipMalloc((void **)&avgSlope,sizeof(float)),"hipMalloc");
    CudaSafe( hipMalloc((void **)&slopesArray,width*sizeof(float)),"hipMalloc");

    int blocksX=(15+width)/16;
    int blocksY=(15+height)/16;
    dim3 blocks(blocksX,blocksY);
    dim3 threads(16,16);

    //Calculo la pendiente para cada una de las rectas verticales, por un ajuste de m�nimos cuadrados
    RemoveVerticalLinearTiltStep1Kernel <<< (int)(width+31)/32, 32 >>> (devicePt,width,height,slopesArray);
    CudaSafeKernel("RemoveVerticalLinearTiltStep1");

    //Calculo la media de todas las pendientes
    RemoveVerticalLinearTiltStep2Kernel <<< 1, 1 >>> (devicePt,width,height,avgSlope,slopesArray);
    CudaSafeKernel("RemoveVerticalLinearTiltStep2");

    //Aplico la correci�n a cada punto en funci�n de esa pendiente media
    RemoveVerticalLinearTiltStep3Kernel <<< blocks, threads >>> (devicePt,width,height,avgSlope);
    CudaSafeKernel("RemoveVerticalLinearTiltStep3");

    CudaSafe( hipFree(avgSlope),"hipFree");
    CudaSafe( hipFree(slopesArray),"hipFree");

}

void SetGroundCUDA(float * devicePt, int width,int height){

    float * avgValue;
    CudaSafe( hipMalloc((void **)&avgValue,sizeof(float)),"hipMalloc");

    int blocksX=(15+width)/16;
    int blocksY=(15+height)/16;
    dim3 blocks(blocksX,blocksY);
    dim3 threads(16,16);

    //Calculo la media de los valores en las l�neas verticales laterales
    SetGroundStep1Kernel <<< 1, 1 >>> (devicePt,width,height,avgValue);
    CudaSafeKernel("SetGroundStep1");

    //Si hay alg�n punto con un valor menor a ese, le asigno ese valor
    SetGroundStep2Kernel <<< blocks, threads >>> (devicePt,width,height,avgValue);
    CudaSafeKernel("SetGroundStep2");

    CudaSafe( hipFree(avgValue),"hipFree");

}

void OffsetAndMultiplyCUDA(float * devicePt, int width,int height,float offset,float factor){

    int blocksX=(15+width)/16;
    int blocksY=(15+height)/16;
    dim3 blocks(blocksX,blocksY);
    dim3 threads(16,16);

    OffsetAndMultiplyKernel <<< blocks, threads >>> (devicePt,width,height,offset,factor);
    CudaSafeKernel("OffsetAndMultiply");

}

void MakeSymmetricCUDA(float * devicePt, int width,int height){

    int * maxPos;
    CudaSafe( hipMalloc((void **)&maxPos,sizeof(int)*height),"hipMalloc");
    float * auxMatrixDevice;
    CudaSafe( hipMalloc((void **)&auxMatrixDevice,sizeof(float)*height*width),"hipMalloc");
    //Hago una copia de los datos
    CudaSafe( hipMemcpy(auxMatrixDevice, devicePt, sizeof(float)*height*width,hipMemcpyDeviceToDevice),"hipMemcpy");

    int blocksX=(15+width)/16;
    int blocksY=(15+height)/16;
    dim3 blocks(blocksX,blocksY);
    dim3 threads(16,16);

    //Calculo las posiciones de los m�ximos para cada recta horizontal
    MakeSymmetricStep1Kernel <<< (int)(height+31)/32, 32 >>> (devicePt,width,height,maxPos);
    CudaSafeKernel("MakeSymmetricStep1");

    //Recoloco la imagen desde la copia, promediando lado izquierdo y derecho respecto al m�ximo y centr�ndolo
    MakeSymmetricStep2Kernel <<< blocks, threads >>> (auxMatrixDevice,devicePt,width,height,maxPos);
    CudaSafeKernel("MakeSymmetricStep2");

    CudaSafe( hipFree(maxPos),"hipFree");
    CudaSafe( hipFree(auxMatrixDevice),"hipFree");

}

void AbelInversionCUDA(float * devicePt, int width,int height, float dx,int neighbours){
    float * derivative;
    CudaSafe( hipMalloc((void **)&derivative,sizeof(float)*height*width),"hipMalloc");

    int blocksX=(15+width)/16;
    int blocksY=(15+height)/16;
    dim3 blocks(blocksX,blocksY);
    dim3 threads(16,16);

    //Calculo la derivada en cada l�nea horizontal
    DerivateKernel<<< blocks, threads >>> (devicePt,derivative,width,height,dx,neighbours);
    CudaSafeKernel("DerivativeKernel");

    //Hago la inversion de abel en cada recta horizontal secuencialmente (Si no tendr�a que reservar much�sima memoria)
    AbelInversionKernel<<< blocks, threads >>> (derivative,devicePt,width,height);
    CudaSafeKernel("AbelInversionKernel");

    CudaSafe( hipFree(derivative),"hipFree");
}

float * CropRectangleCUDA(float *devicePt,int width,int height,int x1,int y1,int x2, int y2){
    float * resultDevice;

    CudaSafe( hipMalloc((void **)&resultDevice,sizeof(float)*(x2-x1+1)*(y2-y1+1)),"hipMalloc");

    int blocksX=(15+width)/16;
    int blocksY=(15+height)/16;

    dim3 blocks(blocksX,blocksY);
    dim3 threads(16,16);
    CropRectangleKernel <<< blocks, threads >>> (devicePt, resultDevice,width,height,x1,y1,x2,y2);
    CudaSafeKernel("CropRectangle");

    return resultDevice;

}

float * GetPhaseLineCUDA(float * devicePt,int width,int height,int x1,int y1,int x2, int y2,int outputSize){

    //Reservo memoria para la salida
    float * output;
    CudaSafe( hipMalloc((void **)&output,sizeof(float)*outputSize),"hipMalloc");

    //Calculo el valor que hay que poner en cada punto de la l�na en funci�n de los valores
    GetPhaseLineKernel <<< (int)(outputSize+31)/32, 32 >>> (devicePt,width,height,x1,y1,x2,y2,output,outputSize);
    CudaSafeKernel("GetPhaseLine");

    //Devuelvo la l�nea
    return output;

}
